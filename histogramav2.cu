#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define N 500000000             //Numero de valores de entrada
#define M 8                     //Tamaño del histograma

#define REPETICONES 10000       //Repeticon de pruevas para calculo de media, max y min
#define SCALA 50                //Datos calculados en cada hilo

__device__ int vector_V[N];     //Vector de datos de entrada
__device__ int vector_H[M];     //Vector del histograma

/**
* Funcion para la comprovacion de errores cuda 
*/
static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
*   Kernel para inicializacion de datos de entrada
*/
__global__ void inicializa_v(int random, hiprandState *states, int threadsPerBlock, int blocksPerGrid){
    int iteraciones= SCALA;
    if(blocksPerGrid-1 == blockIdx.x && threadIdx.x == threadsPerBlock -1){
        iteraciones = iteraciones + (N % SCALA);
    }
    unsigned id_x = blockIdx.x*blockDim.x + threadIdx.x;
    hiprandState *state = states + id_x;

    hiprand_init(random, id_x, 0, state);
    for(int i = 0; i < iteraciones; i++){
        if(id_x*SCALA+i < N){
            vector_V[id_x*SCALA+i] = (int)((hiprand_uniform(state)*1000)) % M;
        }
    }

}

/**
*   Kernel para inicializacion del vector de histograma
*/
__global__ void inicializa_h(){
    unsigned id_x = blockIdx.x*blockDim.x + threadIdx.x;
    vector_H[id_x] = 0;
}
/**
*   Kernel para calculo del histograma
*/
__global__ void histograma(int threadsPerBlock, int blocksPerGrid){
    int vector[M];
    for(int i =0; i < M;i++){
        vector[i] =0;
    }
    int iteraciones= SCALA;
    if(blocksPerGrid-1 == blockIdx.x && threadIdx.x == threadsPerBlock -1){
       iteraciones = iteraciones + (N % SCALA);
    }
    unsigned id_x = blockIdx.x*blockDim.x + threadIdx.x;
    for(int i = 0; i < iteraciones; i++){
        if(id_x*SCALA+i < N){
            int mod = vector_V[id_x*SCALA+i]%M;
            vector[mod]++;
        }
    }
    for(int i =0; i < M;i++){
        int a =vector[i];
        atomicAdd(&vector_H[i],a);
    }
}



int main(){
    srand(time(NULL));
    static hiprandState *states = NULL;
    //int h_v_d[N];
    int h_v_h[M];
    int threadsPerBlock = 1024;
    int blocksPerGrid =((N/SCALA) + threadsPerBlock - 1) / threadsPerBlock;

    float t_duration[REPETICONES];
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for(int j = 0; j< REPETICONES; j++){
        CUDA_CHECK_RETURN(hipEventRecord(start, 0));

        CUDA_CHECK_RETURN(hipMalloc((void **)&states, sizeof(hiprandState) * threadsPerBlock  * blocksPerGrid));
        inicializa_v<<<blocksPerGrid, threadsPerBlock>>>(rand(),states, threadsPerBlock,blocksPerGrid);
        CUDA_CHECK_RETURN(hipGetLastError());
        inicializa_h<<<1,M>>>();
        CUDA_CHECK_RETURN(hipGetLastError());

        histograma<<<blocksPerGrid,threadsPerBlock>>>(threadsPerBlock,blocksPerGrid);
        CUDA_CHECK_RETURN(hipGetLastError());

        //CUDA_CHECK_RETURN(cudaMemcpyFromSymbol(h_v_d, vector_V, N*sizeof(int)));
        CUDA_CHECK_RETURN(hipMemcpyFromSymbol(h_v_h, HIP_SYMBOL(vector_H), M*sizeof(int)));
        int acumula =0;
        for(int  i = 0; i<M; i++){
            std::cout<<h_v_h[i]<<" ";
            acumula += h_v_h[i];
        }
        std::cout<<"\n-------------------------"<<acumula<<"-----------------------------------\n";
        /*
        for(int  i = 0; i<10; i++){
            for(int  j = 0; j<10; j++){
            std::cout<<h_v_d[10*i+j]<<" ";
            };
            std::cout<<"\n";
        }
        */  
        CUDA_CHECK_RETURN(hipFree(states));
        CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
        CUDA_CHECK_RETURN(hipEventSynchronize(stop));

        CUDA_CHECK_RETURN(hipEventElapsedTime(&t_duration[j],start,stop));  
    }
    float t_max =0, t_min= FLT_MAX, media=0;
    for(int i = 0; i< REPETICONES; i++){
        media +=t_duration[i];
        if(t_duration[i] > t_max){
            t_max =t_duration[i]; 
        }
        if(t_duration[i]< t_min){
            t_min= t_duration[i];
        }
    }
    std::cout<< "Se han realizado "<<REPETICONES<<" repeticones\n";
    std::cout<<"Obteniendo de media: "<<media/REPETICONES<<"ms \n";
    std::cout<<"Y de máximo: "<<t_max<<"ms  y mínimo: "<<t_min<<"ms\n";

    return 0;
}



/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {

	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (EXIT_FAILURE);
}
